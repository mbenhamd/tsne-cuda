#include "hip/hip_runtime.h"
/*
    Compute t-SNE via Barnes-Hut for NlogN time.
*/

#include "include/bh_tsne.h"

void tsnecuda::bh::RunTsne(tsnecuda::Options &opt,
                            tsnecuda::GpuOptions &gpu_opt)
{
    // Check the validity of the options file
    if (!opt.validate()) {
        std::cout << "E: Invalid options file. Terminating." << std::endl;
        return;
    }

    // Construct the handles
    hipblasHandle_t dense_handle;
    CublasSafeCall(hipblasCreate(&dense_handle));
    hipsparseHandle_t sparse_handle;
    CusparseSafeCall(hipsparseCreate(&sparse_handle));

    // Set CUDA device properties
    const int num_blocks = gpu_opt.sm_count;
    
    // Construct sparse matrix descriptor
    hipsparseMatDescr_t sparse_matrix_descriptor;
    hipsparseCreateMatDescr(&sparse_matrix_descriptor);
    hipsparseSetMatType(sparse_matrix_descriptor, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(sparse_matrix_descriptor,HIPSPARSE_INDEX_BASE_ZERO);
    
    // Setup some return information if we're working on snapshots
    int snap_interval;
    int snap_num = 0;
    if (opt.return_style == tsnecuda::RETURN_STYLE::SNAPSHOT) {
      snap_interval = opt.iterations / (opt.num_snapshots-1);
    }

    // Get constants from options
    const int num_points = opt.num_points;
    const int num_neighbors = (opt.num_neighbors < num_points) ? opt.num_neighbors : num_points;
    const float *high_dim_points = opt.points;
    float grad_norm;
    const int high_dim = opt.num_dims;
    const float perplexity = opt.perplexity;
    const float perplexity_search_epsilon = opt.perplexity_search_epsilon;
    const float eta = opt.learning_rate;
    float momentum = opt.pre_exaggeration_momentum;
    float attr_exaggeration = opt.early_exaggeration;
    float normalization;

    // Theta governs tolerance for Barnes-Hut recursion
    const float theta = opt.theta;
    const float epsilon_squared = opt.epssq;    
    // Figure out number of nodes needed for BH tree
    int nnodes = num_points * 2;
    if (nnodes < 1024 * num_blocks) nnodes = 1024 * num_blocks;
    while ((nnodes & (gpu_opt.warp_size - 1)) != 0)
        nnodes++;
    nnodes--;
    const int num_nodes = nnodes;
    opt.num_nodes = num_nodes;
    
    // Allocate host memory
    float *knn_squared_distances = new float[num_points * num_neighbors];
    memset(knn_squared_distances, 0, num_points * num_neighbors * sizeof(float));
    long *knn_indices = new long[num_points * num_neighbors];

    // Initialize global variables
    thrust::device_vector<int> err_device(1);
    tsnecuda::bh::Initialize(gpu_opt, err_device);
    // Compute approximate K Nearest Neighbors and squared distances
    tsnecuda::util::KNearestNeighbors(gpu_opt, knn_indices, knn_squared_distances, high_dim_points, high_dim, num_points, num_neighbors);
    thrust::device_vector<long> knn_indices_long_device(knn_indices, knn_indices + num_points * num_neighbors);
    thrust::device_vector<int> knn_indices_device(num_points * num_neighbors);
    tsnecuda::util::PostprocessNeighborIndices(gpu_opt, knn_indices_device, knn_indices_long_device, 
                                                        num_points, num_neighbors);
    // Max-norm the distances to avoid exponentiating by large numbers
    thrust::device_vector<float> knn_squared_distances_device(knn_squared_distances, 
                                            knn_squared_distances + (num_points * num_neighbors));
    tsnecuda::util::MaxNormalizeDeviceVector(knn_squared_distances_device);
    // Search Perplexity
    thrust::device_vector<float> pij_non_symmetric_device(num_points * num_neighbors);
    tsnecuda::bh::SearchPerplexity(gpu_opt, dense_handle, pij_non_symmetric_device, knn_squared_distances_device, 
                                    perplexity, perplexity_search_epsilon, num_points, num_neighbors);
    // Clean up memory
    knn_squared_distances_device.clear();
    knn_squared_distances_device.shrink_to_fit();
    knn_indices_long_device.clear();
    knn_indices_long_device.shrink_to_fit();
    delete[] knn_squared_distances;
    delete[] knn_indices;
    // Symmetrize the pij matrix
    thrust::device_vector<float> sparse_pij_device;
    thrust::device_vector<int> pij_row_ptr_device;
    thrust::device_vector<int> pij_col_ind_device;
    tsnecuda::util::SymmetrizeMatrix(sparse_handle, sparse_pij_device, pij_row_ptr_device,
                                        pij_col_ind_device, pij_non_symmetric_device, knn_indices_device,
                                        opt.magnitude_factor, num_points, num_neighbors);
    const int num_nonzero = sparse_pij_device.size();

    // Clean up memory
    knn_indices_device.clear();
    knn_indices_device.shrink_to_fit();
    pij_non_symmetric_device.clear();
    pij_non_symmetric_device.shrink_to_fit();   
    // Declare memory
    thrust::device_vector<float> pij_x_qij_device(sparse_pij_device.size());
    thrust::device_vector<float> repulsive_forces_device((num_nodes + 1) * 2, 0);
    thrust::device_vector<float> attractive_forces_device(opt.num_points * 2, 0);
    thrust::device_vector<float> gains_device(opt.num_points * 2, 1);
    thrust::device_vector<float> old_forces_device(opt.num_points * 2, 0); // for momentum
    thrust::device_vector<int> cell_starts_device(num_nodes + 1);
    thrust::device_vector<int> children_device((num_nodes + 1) * 4);
    thrust::device_vector<float> cell_mass_device(num_nodes + 1, 1.0); // TODO: probably don't need massl
    thrust::device_vector<int> cell_counts_device(num_nodes + 1);
    thrust::device_vector<int> cell_sorted_device(num_nodes + 1);
    thrust::device_vector<float> normalization_vec_device(num_nodes + 1);
    thrust::device_vector<float> x_max_device(num_blocks * gpu_opt.bounding_kernel_factor);
    thrust::device_vector<float> y_max_device(num_blocks * gpu_opt.bounding_kernel_factor);
    thrust::device_vector<float> x_min_device(num_blocks * gpu_opt.bounding_kernel_factor);
    thrust::device_vector<float> y_min_device(num_blocks * gpu_opt.bounding_kernel_factor);
    thrust::device_vector<float> ones_device(opt.num_points * 2, 1); // This is for reduce summing, etc.
    thrust::device_vector<int> coo_indices_device(sparse_pij_device.size()*2);

    tsnecuda::util::Csr2Coo(gpu_opt, coo_indices_device, pij_row_ptr_device,
                            pij_col_ind_device, num_points, num_nonzero);

    // Initialize Low-Dim Points
    thrust::device_vector<float> points_device((num_nodes + 1) * 2);
    thrust::device_vector<float> random_vector_device(points_device.size());

    std::default_random_engine generator(opt.random_seed);
    std::normal_distribution<double> distribution1(0.0, 1.0);
    thrust::host_vector<float> h_points_device((num_nodes+ 1) * 2);

    // Initialize random noise vector
    for (int i = 0; i < (num_nodes+1)*2; i++) h_points_device[i] = 0.001 * distribution1(generator);
    thrust::copy(h_points_device.begin(), h_points_device.end(), random_vector_device.begin());

    // TODO: this will only work with gaussian init
    if (opt.initialization == tsnecuda::TSNE_INIT::UNIFORM) { // Random uniform initialization
        points_device = tsnecuda::util::RandomDeviceVectorInRange(generator, (num_nodes+1)*2, -5, 5);
    } else if (opt.initialization == tsnecuda::TSNE_INIT::GAUSSIAN) { // Random gaussian initialization
        // Generate some Gaussian noise for the points
        for (int i = 0; i < (num_nodes+ 1) * 2; i++) h_points_device[i] = 0.0001 * distribution1(generator);
        thrust::copy(h_points_device.begin(), h_points_device.end(), points_device.begin());
    } else if (opt.initialization == tsnecuda::TSNE_INIT::RESUME) { // Preinit from vector
        // Load from vector
        if(opt.preinit_data != nullptr) {
          thrust::copy(opt.preinit_data, opt.preinit_data+(num_nodes+1)*2, points_device.begin());
        } else {
          std::cerr << "E: Invalid initialization. Initialization points are null." << std::endl;
          exit(1);
        }
    } else if (opt.initialization == tsnecuda::TSNE_INIT::VECTOR) { // Preinit from vector points only
        // Copy the pre-init data
        if(opt.preinit_data != nullptr) {
          thrust::copy(opt.preinit_data, opt.preinit_data+opt.num_points, points_device.begin());
          thrust::copy(opt.preinit_data+opt.num_points+1, opt.preinit_data+opt.num_points*2 , points_device.begin()+(num_nodes+1));
        } else {
          std::cerr << "E: Invalid initialization. Initialization points are null." << std::endl;
          exit(1);
        }
    } else { // Invalid initialization
        std::cerr << "E: Invalid initialization type specified." << std::endl;
        exit(1);
    }
    
    // Dump file
    float *host_ys = nullptr;
    std::ofstream dump_file;
    if (opt.get_dump_points()) {
        dump_file.open(opt.get_dump_file());
        host_ys = new float[(num_nodes + 1) * 2];
        dump_file << num_points << " " << 2 << std::endl;
    }

    #ifndef NO_ZMQ 
        bool send_zmq = opt.get_use_interactive();
        zmq::context_t context(1);
        zmq::socket_t publisher(context, ZMQ_REQ);
        if (opt.get_use_interactive()) {

        // Try to connect to the socket
            if (opt.verbosity >= 1)
                std::cout << "Initializing Connection...." << std::endl;
            publisher.setsockopt(ZMQ_RCVTIMEO, opt.get_viz_timeout());
            publisher.setsockopt(ZMQ_SNDTIMEO, opt.get_viz_timeout());
            if (opt.verbosity >= 1)
                std::cout << "Waiting for connection to visualization for 10 secs...." << std::endl;
            publisher.connect(opt.get_viz_server());

            // Send the number of points we should be expecting to the server
            std::string message = std::to_string(opt.num_points);
            send_zmq = publisher.send(message.c_str(), message.length());

            // Wait for server reply
            zmq::message_t request;
            send_zmq = publisher.recv (&request);
            
            // If there's a time-out, don't bother.
            if (send_zmq) {
                if (opt.verbosity >= 1)
                    std::cout << "Visualization connected!" << std::endl;
            } else {
                std::cout << "No Visualization Terminal, continuing..." << std::endl;
                send_zmq = false;
            }
        }
    #else
        if (opt.get_use_interactive()) 
            std::cout << "This version is not built with ZMQ for interative viz. Rebuild with WITH_ZMQ=TRUE for viz." << std::endl;
    #endif

    // Support for infinite iteration
    for (size_t step = 0; step != opt.iterations; step++) {

        // Setup learning rate schedule
        if (step == opt.force_magnify_iters) {
            momentum = opt.post_exaggeration_momentum;
            attr_exaggeration = 1.0f;
        }
        // Compute Bounding Box
        tsnecuda::bh::ComputeBoundingBox(gpu_opt,
                                         cell_starts_device,
                                         children_device,
                                         cell_mass_device,
                                         points_device,
                                         x_max_device,
                                         y_max_device,
                                         x_min_device,
                                         y_min_device,
                                         num_nodes, 
                                         num_points, 
                                         num_blocks);
        // Tree Builder
        tsnecuda::bh::BuildTree(gpu_opt,
                                err_device,
                                children_device,
                                cell_starts_device,
                                cell_mass_device,
                                points_device,
                                num_nodes,
                                num_points,
                                num_blocks);

        // Tree Summarization
        tsnecuda::bh::SummarizeTree(gpu_opt,
                                    cell_counts_device,
                                    children_device,
                                    cell_mass_device,
                                    points_device,
                                    num_nodes,
                                    num_points,
                                    num_blocks);

        // Sort By Morton Code
        tsnecuda::bh::SortCells(gpu_opt,
                                cell_sorted_device,
                                cell_starts_device,
                                children_device,
                                cell_counts_device,
                                num_nodes,
                                num_points,
                                num_blocks);

        // Calculate Repulsive Forces
        tsnecuda::bh::ComputeRepulsiveForces(gpu_opt,
                                             err_device,
                                             repulsive_forces_device,
                                             normalization_vec_device,
                                             cell_sorted_device,
                                             children_device,
                                             cell_mass_device,
                                             points_device,
                                             theta, 
                                             epsilon_squared,
                                             num_nodes,
                                             num_points,
                                             num_blocks);
        // Compute normalization
        normalization = thrust::reduce(normalization_vec_device.begin(),
                                        normalization_vec_device.end(),
                                        0.0f,
                                        thrust::plus<float>());

        // Calculate Attractive Forces
        tsnecuda::bh::ComputeAttractiveForces(gpu_opt,
                                              sparse_handle,
                                              sparse_matrix_descriptor,
                                              attractive_forces_device,
                                              pij_x_qij_device,
                                              sparse_pij_device,
                                              pij_row_ptr_device,
                                              pij_col_ind_device,
                                              coo_indices_device,
                                              points_device,
                                              ones_device,
                                              num_nodes,
                                              num_points,
                                              num_nonzero);

        // Apply Forces
        tsnecuda::bh::ApplyForces(gpu_opt,
                                  points_device,
                                  attractive_forces_device,
                                  repulsive_forces_device,
                                  gains_device,
                                  old_forces_device,
                                  eta,
                                  normalization,
                                  momentum,
                                  attr_exaggeration,
                                  num_nodes,
                                  num_points,
                                  num_blocks);
        // Add a bit of random motion to prevent points from being on top of each other
        thrust::transform(points_device.begin(), points_device.end(), random_vector_device.begin(),
                            points_device.begin(), thrust::plus<float>());
        // Compute the gradient norm
        tsnecuda::util::SquareDeviceVector(attractive_forces_device, old_forces_device);
        thrust::transform(attractive_forces_device.begin(), attractive_forces_device.begin()+num_points, 
                          attractive_forces_device.begin()+num_points, attractive_forces_device.begin(), 
                          thrust::plus<float>());                       
        tsnecuda::util::SqrtDeviceVector(attractive_forces_device, attractive_forces_device);                      
        grad_norm = thrust::reduce(attractive_forces_device.begin(), attractive_forces_device.begin()+num_points, 
                                         0.0f, thrust::plus<float>()) / num_points;
        thrust::fill(attractive_forces_device.begin(), attractive_forces_device.end(), 0.0f);
        if (grad_norm < opt.min_gradient_norm) {
            if (opt.verbosity >= 1) std::cout << "Reached minimum gradient norm: " << opt.min_gradient_norm << std::endl;
            break;
        }

        if (opt.verbosity >= 1 && step % opt.print_interval == 0) {
            std::cout << "[Step " << step << "] Avg. Gradient Norm: " << grad_norm << std::endl;
        }
            
        
        #ifndef NO_ZMQ
            if (send_zmq) {
            zmq::message_t message(sizeof(float)*opt.num_points*2);
            thrust::copy(points_device.begin(), points_device.begin()+opt.num_points, static_cast<float*>(message.data()));
            thrust::copy(points_device.begin()+num_nodes+1, points_device.begin()+num_nodes+1+opt.num_points, static_cast<float*>(message.data())+opt.num_points);
            bool res = false;
            res = publisher.send(message);
            zmq::message_t request;
            res = publisher.recv(&request);
            if (!res) {
                std::cout << "Server Disconnected, Not sending anymore for this session." << std::endl;
            }
            send_zmq = res;
            }
        #endif

        if (opt.get_dump_points() && step % opt.get_dump_interval() == 0) {
            thrust::copy(points_device.begin(), points_device.end(), host_ys);
            for (int i = 0; i < opt.num_points; i++) {
                dump_file << host_ys[i] << " " << host_ys[i + num_nodes + 1] << std::endl;
            }
        }

        // Handle snapshoting
        if (opt.return_style == tsnecuda::RETURN_STYLE::SNAPSHOT && step % snap_interval == 0 && opt.return_data != nullptr) {
          thrust::copy(points_device.begin(),
                       points_device.begin()+opt.num_points, 
                       snap_num*opt.num_points*2 + opt.return_data);
          thrust::copy(points_device.begin()+num_nodes+1, 
                       points_device.begin()+num_nodes+1+opt.num_points,
                       snap_num*opt.num_points*2 + opt.return_data+opt.num_points);
          snap_num += 1;
        }

    }

    // Clean up the dump file if we are dumping points
    if (opt.get_dump_points()){
      delete[] host_ys;
      dump_file.close();
    }

    // Handle a once off return type
    if (opt.return_style == tsnecuda::RETURN_STYLE::ONCE && opt.return_data != nullptr) {
      thrust::copy(points_device.begin(), points_device.begin()+opt.num_points, opt.return_data);
      thrust::copy(points_device.begin()+num_nodes+1, points_device.begin()+num_nodes+1+opt.num_points, opt.return_data+opt.num_points);
    }

    // Handle snapshoting
    if (opt.return_style == tsnecuda::RETURN_STYLE::SNAPSHOT && opt.return_data != nullptr) {
      thrust::copy(points_device.begin(), points_device.begin()+opt.num_points, snap_num*opt.num_points*2 + opt.return_data);
      thrust::copy(points_device.begin()+num_nodes+1, points_device.begin()+num_nodes+1+opt.num_points, snap_num*opt.num_points*2 + opt.return_data+opt.num_points);
    }

    opt.grad_norm = &grad_norm;
    // Return some final values
    opt.trained = true;
    opt.trained_norm = normalization;

    return;
}

